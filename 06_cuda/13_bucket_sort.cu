
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <vector>

__global__ void scan(int *a, int *b, int range) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  for (int j=1; j<range; j<<=1) {
    b[i] = a[i];
    __syncthreads();
    if (i >= j) a[i] += b[i-j];
    __syncthreads();
  }
}
__global__ void set_bucket(int *bucket, int *key) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  atomicAdd(&bucket[key[i]], 1);
}

__global__ void set_offset(int *offset, int *bucket) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= 1) offset[i] = bucket[i-1];
}
__global__ void set_key(int *key, int *offset, int *bucket) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = offset[i];
  for (; bucket[i]>0; bucket[i]--) {
    key[j++] = i;
  }
}

int main() {
  // both n and range should be no more than 1024
  const int n = 50;
  const int range = 128;

  int *key, *bucket, *offset, *tmp;

  hipMallocManaged(&key, n*sizeof(int));
  hipMallocManaged(&bucket, range*sizeof(int));
  hipMallocManaged(&offset, range*sizeof(int));
  hipMallocManaged(&tmp, range*sizeof(int));

  // init key
  for (int i=0; i<n; i++) {
    key[i] = rand() % range;
    printf("%d ",key[i]);
  }
  printf("\n");

  // set bucket
  hipMemset(bucket, 0, range*sizeof(int));
  set_bucket<<<1, n>>>(bucket, key);
  hipDeviceSynchronize();

  // set offset
  hipMemset(offset, 0, range*sizeof(int));
  set_offset<<<1, range>>>(offset, bucket);
  // set tmp
  hipMemset(tmp, 0, range*sizeof(int));
  hipDeviceSynchronize();
  scan<<<1, range>>>(offset, tmp, range);
  hipDeviceSynchronize();

  // set key
  set_key<<<1, range>>>(key, offset, bucket);
  hipDeviceSynchronize();


  for (int i=0; i<n; i++) {
    printf("%d ",key[i]);
  }
  printf("\n");

  hipFree(key);
  hipFree(bucket);
  hipFree(offset);
  hipFree(tmp);
}
